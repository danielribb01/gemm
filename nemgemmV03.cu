#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <cuda/barrier>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <ctime>
#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_bf16.h>
#include <cassert>

// binding libraries
#include <torch/extension.h>
#include <pybind11/pybind11.h>

namespace cde = cuda::device::experimental;
typedef __hip_bfloat16 bf16;
using barrier = cuda::barrier<cuda::thread_scope_block>;

#define CEIL_DIV(x, y) (((x) + (y) - 1) / (y))

// Global variables for TMA maps to avoid reallocation
static CUtensorMap *d_tma_map_A = nullptr;
static CUtensorMap *d_tma_map_B = nullptr;
static constexpr int maxCollums = 512;
static int prev_m = 0, prev_n = 0, prev_k = 0;

template <int BlockMajorSize, int BlockMinorSize>
void create_tensor_map(CUtensorMap *tma_map, bf16* data_ptr, int blocks_height, int blocks_width) {
    void* gmem_address = static_cast<void*>(data_ptr);
    uint64_t gmem_prob_shape[5] = {
        static_cast<uint64_t>(BlockMinorSize * blocks_width), 
        static_cast<uint64_t>(BlockMajorSize * blocks_height), 
        1, 1, 1
    };
    uint64_t gmem_prob_stride[5] = {
        sizeof(bf16), 
        sizeof(bf16) * BlockMinorSize * blocks_width, 
        0, 0, 0
    };
    uint32_t smem_box_shape[5] = {
        static_cast<uint32_t>(BlockMinorSize), 
        static_cast<uint32_t>(BlockMajorSize), 
        1, 1, 1
    };
    uint32_t smem_box_stride[5] = {1, 1, 1, 1, 1};

    hipError_t result = cuTensorMapEncodeTiled(
        tma_map, CU_TENSOR_MAP_DATA_TYPE_BFLOAT16, 2, gmem_address, gmem_prob_shape,
        gmem_prob_stride + 1, smem_box_shape, smem_box_stride, CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_128B, CU_TENSOR_MAP_L2_PROMOTION_L2_128B, CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE);

    assert(result == hipSuccess);
}

template <int BlockMajorSize, int BlockMinorSize>
__host__ static inline CUtensorMap* allocate_and_create_tensor_map(bf16* data_ptr, int blocks_height, int blocks_width) {
    CUtensorMap *tma_map_d_tmp;
    hipMalloc(&tma_map_d_tmp, sizeof(CUtensorMap));
    CUtensorMap tma_map_host;
    create_tensor_map<BlockMajorSize, BlockMinorSize>(&tma_map_host, data_ptr, blocks_height, blocks_width);
    hipMemcpy(tma_map_d_tmp, &tma_map_host, sizeof(CUtensorMap), hipMemcpyHostToDevice);
    return tma_map_d_tmp;
}

__device__ static inline void tmem_alloc_maxColumns(uint32_t* tmem_base_addr) {
    uint32_t tmem_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(tmem_base_addr));
    asm volatile (
        "tcgen05.alloc.cta_group::1.sync.aligned.shared::cta.b32 [%0], %1;\n"
        :
        : "r"(tmem_ptr), "r"(maxCollums));
}

__device__ static inline void dealloc_tmem(uint32_t tmem_base_addr) {
    asm volatile(
        "{\n\t"
      "tcgen05.dealloc.cta_group::1.sync.aligned.b32  %0, %1; \n\t"
      "}"
      :
      : "r"(tmem_base_addr), "r"(maxCollums));
}

__device__ static inline void release_lock() {
    asm volatile("tcgen05.relinquish_alloc_permit.cta_group::1.sync.aligned;\n" ::);
}


__device__ static inline uint64_t matrix_descriptor_encode(uint64_t x) { 
    return (((x) & 0x3FFFF) >> 0x4); 
}



__device__ void cta_commit(uint64_t &mma_barrier_addr) {
    uint32_t mma_barrier_ptr =  static_cast<uint32_t>(__cvta_generic_to_shared(&mma_barrier_addr));
    asm volatile(
        "tcgen05.commit.cta_group::1.mbarrier::arrive::one.b64 [%0];\n" 
        :: "r"(mma_barrier_ptr) : "memory"
    );
}

__device__ static inline void barrier_init(uint64_t &mma_barrier_addr) {
    uint32_t mma_barrier_ptr =  static_cast<uint32_t>(__cvta_generic_to_shared(&mma_barrier_addr));
    asm volatile(
        "mbarrier.init.shared::cta.b64 [%0], %1;\n"
        :
        : "r"(mma_barrier_ptr), "n"(128)
    );
}

__device__ static inline void barrier_arrive(uint64_t &mma_barrier_addr) {
    uint32_t mma_barrier_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(&mma_barrier_addr));
    asm volatile(
        "mbarrier.arrive.shared::cta.b64 _,[%0];\n"
        :
        : "r"(mma_barrier_ptr)
    );
}

__device__ static inline void load_wait() {
    asm volatile (
        "tcgen05.wait::ld.sync.aligned;\n"
    );
}

// Load accumulator from TMEM to registers using tcgen05.ld
__device__ void load_tmem_to_registers(float d[32][64], uint32_t const &tmem_base_addr) {
    asm volatile (" {\n"
                  " tcgen05.ld.sync.aligned.32x32b.x64.b32 "
                  "{%0, %1, %2, %3, "
                  " %4, %5, %6, %7, "
                  " %8, %9, %10, %11, "
                  " %12, %13, %14, %15, "
                  " %16, %17, %18, %19, "
                  " %20, %21, %22, %23, "
                  " %24, %25, %26, %27, "
                  " %28, %29, %30, %31, "
                  " %32, %33, %34, %35, "
                  " %36, %37, %38, %39, "
                  " %40, %41, %42, %43, "
                  " %44, %45, %46, %47, "
                  " %48, %49, %50, %51, "
                  " %52, %53, %54, %55, "
                  " %56, %57, %58, %59, "
                  " %60, %61, %62, %63}, "              
                  "[%64];\n"
                  "}\n"
                : "=f"(d[0][0]), "=f"(d[0][1]), "=f"(d[0][2]), "=f"(d[0][3]),
                  "=f"(d[0][4]), "=f"(d[0][5]), "=f"(d[0][6]), "=f"(d[0][7]),
                  "=f"(d[0][8]), "=f"(d[0][9]), "=f"(d[0][10]), "=f"(d[0][11]),
                  "=f"(d[0][12]), "=f"(d[0][13]), "=f"(d[0][14]), "=f"(d[0][15]),
                  "=f"(d[0][16]), "=f"(d[0][17]), "=f"(d[0][18]), "=f"(d[0][19]),
                  "=f"(d[0][20]), "=f"(d[0][21]), "=f"(d[0][22]), "=f"(d[0][23]),
                  "=f"(d[0][24]), "=f"(d[0][25]), "=f"(d[0][26]), "=f"(d[0][27]),
                  "=f"(d[0][28]), "=f"(d[0][29]), "=f"(d[0][30]), "=f"(d[0][31]),
                  "=f"(d[0][32]), "=f"(d[0][33]), "=f"(d[0][34]), "=f"(d[0][35]),
                  "=f"(d[0][36]), "=f"(d[0][37]), "=f"(d[0][38]), "=f"(d[0][39]),
                  "=f"(d[0][40]), "=f"(d[0][41]), "=f"(d[0][42]), "=f"(d[0][43]),
                  "=f"(d[0][44]), "=f"(d[0][45]), "=f"(d[0][46]), "=f"(d[0][47]),
                  "=f"(d[0][48]), "=f"(d[0][49]), "=f"(d[0][50]), "=f"(d[0][51]),
                  "=f"(d[0][52]), "=f"(d[0][53]), "=f"(d[0][54]), "=f"(d[0][55]),
                  "=f"(d[0][56]), "=f"(d[0][57]), "=f"(d[0][58]), "=f"(d[0][59]),
                  "=f"(d[0][60]), "=f"(d[0][61]), "=f"(d[0][62]), "=f"(d[0][63])
                :  "r"(tmem_base_addr));
}


template<uint8_t Acc>
__device__ void mma64x64x16(bf16* sA, bf16* sB, uint32_t const &base_tmem_ptr) {
    uint64_t desc_a = 0x4000004000000000 | 
        (matrix_descriptor_encode(static_cast<uint64_t>(__cvta_generic_to_shared(sA))));
    uint64_t desc_b = 0x4000004000000000 | 
        (matrix_descriptor_encode(static_cast<uint64_t>(__cvta_generic_to_shared(sB))));
    
    constexpr uint32_t instruction_desc = 0x04100490;
    constexpr uint32_t mask[4] = {0, 0, 0, 0};
      asm volatile(
        "{\n\t"
        ".reg .pred p;\n\t"
        "setp.ne.b32 p, %4, 0;\n\t"
        "tcgen05.mma.cta_group::1.kind::f16 [%0], %1, %2, %3, {%5, %6, %7, %8}, p; \n\t"
        "}\n"
        :
        : "r"(base_tmem_ptr), "l"(desc_a), "l"(desc_b), "r"(instruction_desc), "r"(Acc),
          "r"(mask[0]), "r"(mask[1]), "r"(mask[2]), "r"(mask[3]));
}

template<int BM, int BN, int BK, int MMA_M, int MMA_N, int MMA_K, int NUM_THREADS>
__global__ void __launch_bounds__(NUM_THREADS) gemm_kernel(
    int M, int N, int K, 
    bf16* C,
    const CUtensorMap* tensorMapA, 
    const CUtensorMap* tensorMapB) {
    int tid = threadIdx.x;
    int warp = tid / 32;

    // Shared memory buffers - 128-byte aligned
    __shared__ alignas(128) bf16 sA[BM * BK];
    __shared__ alignas(128) bf16 sB[BK * BN];
    __shared__ alignas(16) uint32_t tmem_base_addr;
    __shared__ alignas(16) uint64_t mma_barrier_addr;


    // Allocate tensor memory
    if (warp == 0) {
        tmem_alloc_maxColumns(&tmem_base_addr);
    }
    __syncthreads();

    // Output accumulator
    float d[64][64] = {};

    // Block indices
    const int num_blocks_k = K / BK;
    int num_blocks_n = blockIdx.x % (N / BN);
    int num_blocks_m = blockIdx.x / (N / BN);

    // TMA barriers
    __shared__ barrier barA;
    __shared__ barrier barB;

    if (threadIdx.x == 0) {
        init(&barA, blockDim.x);
        init(&barB, blockDim.x);
        cde::fence_proxy_async_shared_cta();
    }
    __syncthreads();

    // Main computation loop
    barrier::arrival_token tokenA, tokenB;
    for (int bkIdx = 0; bkIdx < num_blocks_k; ++bkIdx) {
        // TMA loads
        if (threadIdx.x == 0) {
            cde::cp_async_bulk_tensor_2d_global_to_shared(
                &sA[0], tensorMapA, bkIdx * BK, num_blocks_m * BM, barA);
            tokenA = cuda::device::barrier_arrive_tx(barA, 1, sizeof(sA));
            
            cde::cp_async_bulk_tensor_2d_global_to_shared(
                &sB[0], tensorMapB, bkIdx * BK, num_blocks_n * BN, barB);
            tokenB = cuda::device::barrier_arrive_tx(barB, 1, sizeof(sB));
        } else {
            tokenA = barA.arrive();
            tokenB = barB.arrive();
        }

        barA.wait(std::move(tokenA));
        barB.wait(std::move(tokenB));
        __syncthreads();

    }
    if (tid == 0) {
        barrier_init(mma_barrier_addr); // Initialize barriers

        // Perform MMA operations for different K iterations
        mma64x64x16<0>(&sA[0], &sB[0], tmem_base_addr);
        mma64x64x16<1>(&sA[MMA_K], &sB[MMA_K * BN], tmem_base_addr);
        mma64x64x16<1>(&sA[2 * MMA_K], &sB[2 * MMA_K * BN], tmem_base_addr);
        mma64x64x16<1>(&sA[3 * MMA_K], &sB[2 * MMA_K * BN], tmem_base_addr);
    }

    barrier_arrive(mma_barrier_addr);
    cta_commit(mma_barrier_addr);

    // Load accumulator from TMEM
    if(warp == 0) {
        load_tmem_to_registers(d, tmem_base_addr);
        load_tmem_to_registers(d+32, (tmem_base_addr + 0x00200000));
    }
    load_wait();
        
    bf16 *block_C = C + num_blocks_n * BN * M + num_blocks_m * BM;             
    int idx = tid % 64;
    for(int cols = 0; cols < 32; ++cols) {
        if(warp == 0 || warp == 1) {
            block_C[idx] = __float2bfloat16(d[idx][cols]);
        } else {
            block_C[idx] = __float2bfloat16(d[idx][cols + 32]);
        }
        __syncthreads();
    }
    if(warp == 0) {
        release_lock();
        dealloc_tmem(tmem_base_addr);
    }
    
}

void neoGemmV1(torch::Tensor A, torch::Tensor B, torch::Tensor C) {
    int M = A.size(0); // A -> MxK
    int K = A.size(1);
    int N = B.size(0); // B -> NxK

    // Get data pointers
    bf16* bf16_data_ptr_A = reinterpret_cast<bf16*>(A.data_ptr<at::BFloat16>());
    bf16* bf16_data_ptr_B = reinterpret_cast<bf16*>(B.data_ptr<at::BFloat16>());
    bf16* bf16_data_ptr_C = reinterpret_cast<bf16*>(C.data_ptr<at::BFloat16>());

    // Tile sizes
    constexpr int BM = 64;
    constexpr int BN = 64;
    constexpr int BK = 64;

    // Check if we need to reallocate TMA maps
    if (!d_tma_map_A || M != prev_m || N != prev_n || K != prev_k) {
        // Free previous maps if they exist
        if (d_tma_map_A) hipFree(d_tma_map_A);
        if (d_tma_map_B) hipFree(d_tma_map_B);
        
        // Allocate new TMA maps
        d_tma_map_A = allocate_and_create_tensor_map<BM, BK>(bf16_data_ptr_A, M / BM, K / BK);
        d_tma_map_B = allocate_and_create_tensor_map<BN, BK>(bf16_data_ptr_B, N / BN, K / BK);
        
        prev_m = M;
        prev_n = N;
        prev_k = K;
    }

    // Assert dimensions are correct
    assert(M == prev_m && N == prev_n && K == prev_k);
    assert(M % BM == 0 && N % BN == 0 && K % BK == 0);

    // Launch configuration
    constexpr int NUM_THREADS = 128;
    dim3 grid((M/BM) * (N/BN));
    dim3 block(NUM_THREADS);

    // Launch kernel
    gemm_kernel<BM, BN, BK, 64, 64, 16, NUM_THREADS><<<grid, block>>>(
        M, N, K, bf16_data_ptr_C, d_tma_map_A, d_tma_map_B);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
    }
}

namespace py = pybind11;

// Python binding
PYBIND11_MODULE(neoGEMM, m) {
    m.def("neoGemmV1", &neoGemmV1, "Optimized GEMM with TMA and tcgen05",
          py::arg("A"), py::arg("B"), py::arg("C"));
}
